//Elapsed Real Time for input-4.txt: 0.603, 0.606, 0.599
//Elapsed Real Time for input-5.txt: 1.419, 1.391, 1.408

#include <stdio.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

// Representation for a 2D point with integer coordinates.
typedef struct {
  // Coordinates of the point.
  int x, y;
} Point;
// Return the squared distance between point i and point j in the ptList.
__device__ double distSquared(Point *devPtList, int i, int j ) {
  double dx = (double)devPtList[ i ].x - (double)devPtList[ j ].x;
  double dy = (double)devPtList[ i ].y - (double)devPtList[ j ].y;
  return dx * dx + dy * dy;
}
/**
* This function finds the largest of 3 numbers.
*@param p0 number 1
*@param p1 number 2
*@param p2 number 3
*@return the largest of the 3 parameters
*/
__device__ double findLargest( double p0, double p1, double p2){
    if ( p0 >= p1 && p0 >= p2 ){
         return p0;
    } else if (p1 >= p2 && p1 >= p0) {
        return p1;
    } else
        return p2;
}
/**
* This function finds the smallest of 3 numbers.
*@param p0 number 1
*@param p1 number 2
*@param p2 number 3
*@return the smallest of the 3 parameters
*/
__device__ double findSmallest( double p0, double p1, double p2){
    if ( p0 <= p1 && p0 <= p2 ){
        return p0;
    }else if( p1 <= p2 && p1 <= p0){
        return p1;
    }else
        return p2;
}
/**
* This function determines whether the 3 points are within an acceptable range of each other
* in order to qualify as points of a "semilateral" triangle. That range being +/- 10%.
*@param d0 distance b/w point 1 and point 2
*@param d1 distance b/w point 2 and point 3
*@param d2 distance b/w point 3 and point 1
*@return true/false: true if the difference is acceptable. false if not.
*/
__device__ bool check( double d0, double d1, double d2 ){
    double largest = findLargest( d0, d1, d2 );
    double smallest = findSmallest( d0, d1, d2 );
    double diff = (smallest * .10);
  //  printf("Largest: %.2f, Smallest: %.2f, pts[%.2f, %.2f, %.2f]\n", largest, smallest, d0, d1, d2);
    if ( ( largest - smallest ) < diff ){
        return true;
    } else
        return false;
}
__global__ void countTriangles( int *devCountList, Point *devPtList, int n ) {
  //for(int k = 0; k < n; k++)
    // printf("%d, %d\n", devPtList[k].x, devPtList[k].y);
  // Unique index for this worker, it's the index of the first point
  // in any triangles we're supposed to find.
  int a = blockDim.x * blockIdx.x + threadIdx.x;
  int total = 0;
  // Make sure I actually have something to work on.
  if ( a < n ) {
       //If the index is above 2, then we can check the points.
       double p0, p1, p2;
       //The loop starts comparing : arr[a], arr[i], arr[j]
       //So it effectively compares a to all following indicies.
       for( int i = a + 1; i < n - 1; i++ ){
         for( int j = i + 1; j < n; j++ ){
           p0 = (double)distSquared(devPtList, a, i);
           p0 = sqrt(p0);
           p1 = (double)distSquared(devPtList, i, j);
           p1 = sqrt(p1);
           p2 = (double)distSquared(devPtList, a, j);
           p2 = sqrt(p2);
           if ( check( p0, p1, p2 ) )
               total++;
         }
       }
       //printf("a:%d, pts[%.2f, %.2f, %.2f]\n", a, p0, p1, p2);

  }
  devCountList[a] = total;
}


// List of all points from the input
Point *ptList;
int *countList;
// Number of points in the input.
int ptCount = 0;
// Read the list of all points
void readPoints() {
  // Use a resizable array, increasing capacity as we read
  int capacity = 10;
  ptList = (Point *) malloc( capacity * sizeof( Point ) );
  // Read points until we can't read any more.
  int x, y;
  while ( scanf( "%d%d", &x, &y ) == 2 ) {
    // Grow the point list if needed.
    if ( ptCount >= capacity ) {
      capacity *= 2;
      ptList = (Point *) realloc( ptList, capacity * sizeof( Point ) );
    }
    // Add this new point to the end of the list.
    ptList[ ptCount ].x = x;
    ptList[ ptCount ].y = y;
    ptCount += 1;
  }
}

// General function to report a failure and exit.
static void fail( char const *message ) {
  fprintf( stderr, "%s\n", message );
  exit( 1 );
}

int main( ) {
  readPoints();
  //for(int k = 0; k < ptCount; k++)
    // printf("%d, %d\n", ptList[k].x, ptList[k].y);
  // ...
  countList = (int *)malloc( ptCount * sizeof(int) );
  //
  //
  Point *devPtList = NULL;
  if ( hipMalloc((void **)&devPtList, ptCount * sizeof(Point) ) != hipSuccess )
    fail( "Failed to allocate space for lenght list on device" );
  //copy list to device from host
  if ( hipMemcpy( devPtList, ptList, ptCount * sizeof(Point),
                    hipMemcpyHostToDevice) != hipSuccess )
    fail( "Can't copy list to device from host" );
  int *devCountList = NULL;
  if ( hipMalloc((void **)&devCountList, ptCount * sizeof(int) ) != hipSuccess )
    fail( "Failed to allocate space for lenght list on device" );
  // Block and grid dimensions.
  int threadsPerBlock = 250;
  // Round up.
  int blocksPerGrid = ( ptCount + threadsPerBlock - 1 ) / threadsPerBlock;
  // Run our kernel on these block/grid dimensions
  countTriangles<<<blocksPerGrid, threadsPerBlock>>>( devCountList, devPtList, ptCount );
  if ( hipGetLastError() != hipSuccess )
    fail( "Failure in CUDA kernel execution." );
    // copy list from device to host
    if ( hipMemcpy( ptList, devPtList, ptCount * sizeof(Point),
                     hipMemcpyDeviceToHost) != hipSuccess )
      fail( "Can't copy list from device to host" );
    // copy list from device to host
    if ( hipMemcpy( countList, devCountList, ptCount * sizeof(int),
                     hipMemcpyDeviceToHost) != hipSuccess )
      fail( "Can't copy list from device to host" );
  // Print out a sample of the resulting values.
  int sum = 0;
  for ( int i = 0; i < ptCount; i++ ){
       sum += countList[ i ];
  }
  printf("Triangles: %d\n", sum);
  //
  hipFree( devCountList );
  hipFree( devPtList );
  //
  free( ptList );
  free( countList );

  hipDeviceReset();
}
